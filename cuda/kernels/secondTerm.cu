#include "kernels/utills.h"

/* use cuda to optimize calculation of second term, use multi-stream for further accelaration */
void secondTermCuda(const float *img, float *result, int size, float alpha, float power)
{
    float *img_d{nullptr}, *result_d{nullptr};
    size_t imgSize{size * sizeof(float)};
    printIfCudaFailed(hipMalloc(&img_d, imgSize));
    printIfCudaFailed(hipMemcpy(img_d, img, imgSize, hipMemcpyHostToDevice));
    printIfCudaFailed(hipMalloc(&result_d, imgSize));

    int threadsPerBlock{256};
    int numBlocks(size / 256);
    int remain = size % 256;
    secondTermKernel<<<numBlocks, threadsPerBlock>>>(img_d, result_d, alpha, power);
    printIfCudaFailed(hipMemcpy(result, result_d, imgSize, hipMemcpyDeviceToHost));

    secondTermRemain(img, result, alpha, power, remain, size);

    hipFree(img_d);
    hipFree(result_d);

    return;
}

void secondTermCudaAsync(const float *img, float *result, int size, float alpha, float power)
{
    float *img_d{nullptr}, *result_d{nullptr};
    size_t imgSize{size * sizeof(float)};
    printIfCudaFailed(hipMalloc(&img_d, imgSize));
    printIfCudaFailed(hipMemcpyAsync(img_d, img, imgSize, hipMemcpyHostToDevice));
    printIfCudaFailed(hipMalloc(&result_d, imgSize));

    int itemPerStream{size / 8};
    hipStream_t stream[8];
    for (int i = 0; i < 8; ++i)
        hipStreamCreate(&stream[i]);
    for (int i = 0; i < 8; i++)
    {
        printIfCudaFailed(hipMemcpyAsync(img_d + itemPerStream * i, img + itemPerStream * i,
                                          itemPerStream * sizeof(float), hipMemcpyHostToDevice));
        int threadsPerBlock{256};
        int numBlocks(itemPerStream / 256);
        int remain = itemPerStream % 256;
        secondTermKernel<<<numBlocks, threadsPerBlock>>>(img_d + itemPerStream * i, result_d + itemPerStream * i, alpha, power);
        printIfCudaFailed(hipMemcpyAsync(result, result_d, itemPerStream * sizeof(float), hipMemcpyDeviceToHost));
        secondTermRemain(img + itemPerStream * i, result + itemPerStream * i, alpha, power, remain, size);
    }

    hipFree(img_d);
    hipFree(result_d);

    return;
}