#include "kernels/utills.h"

/* use cuda to optimize array power operation, use multiple stream for acceleration */
void arrayPowerCuda(const float *img, float *result, const int size, const float power)
{
    float *img_d{nullptr}, *result_d{nullptr};
    size_t imgSize{size * sizeof(float)};
    printIfCudaFailed(hipMalloc(&img_d, imgSize));
    printIfCudaFailed(hipMemcpy(img_d, img, imgSize, hipMemcpyHostToDevice));
    printIfCudaFailed(hipMalloc(&result_d, imgSize));

    int threadsPerBlock{256};
    int numBlocks(size / 256);
    int remain = size % 256;
    powerKernel<<<numBlocks, threadsPerBlock>>>(img_d, result_d, power);
    printIfCudaFailed(hipMemcpy(result, result_d, imgSize, hipMemcpyDeviceToHost));
    powerRemain(img, result, power, remain, size);

    hipFree(img_d);
    hipFree(result_d);

    return;
}

/* use cuda to optimize array power operation, use multiple stream for acceleration */
void arrayPowerCudaAsync(const float *img, float *result, const int size, const float power)
{
    float *img_d{nullptr}, *result_d{nullptr};
    size_t imgSize{size * sizeof(float)};
    printIfCudaFailed(hipMalloc(&img_d, imgSize));
    printIfCudaFailed(hipMemcpyAsync(img_d, img, imgSize, hipMemcpyHostToDevice));
    printIfCudaFailed(hipMalloc(&result_d, imgSize));

    int itemPerStream{size / 8};
    hipStream_t stream[8];
    for (int i = 0; i < 8; ++i)
        hipStreamCreate(&stream[i]);
    for (int i = 0; i < 8; i++)
    {
        printIfCudaFailed(hipMemcpyAsync(img_d + itemPerStream * i, img + itemPerStream * i,
                                          itemPerStream * sizeof(float), hipMemcpyHostToDevice));
        int threadsPerBlock{256};
        int numBlocks(itemPerStream / 256);
        int remain = itemPerStream % 256;
        powerKernel<<<numBlocks, threadsPerBlock>>>(img_d + itemPerStream * i, result_d + itemPerStream * i, power);
        printIfCudaFailed(hipMemcpyAsync(result, result_d, itemPerStream * sizeof(float), hipMemcpyDeviceToHost));
        powerRemain(img + itemPerStream * i, result + itemPerStream * i, power, remain, itemPerStream);
    }

    hipFree(img_d);
    hipFree(result_d);

    return;
}