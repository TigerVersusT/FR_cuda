#include "hip/hip_runtime.h"
#include "../utills.h"
#include <iostream>
#include <string>
#include <vector>
#include <omp.h>

using namespace std;

// use two dimensional grid and two dimensional block
__global__ void update_l_curr(float *l_curr, float *delta_l, const int row,
                              const int column, const int n)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    if (c < column && r < row)
    {
        int idx1 = r * column + c;
        int idx2 = (r + n) * (column + 2 * n) + c + n;
        l_curr[idx2] += delta_l[idx1];
    }
}

// use one dimensional grid and one dimensional block, exist warp divergence
__global__ void imageRelpaceWith(float *img, const float threshold, const int size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < size)
    {
        if (img[x] < threshold)
        {
            img[x] = threshold;
        }
    }
}

// use one dimensional grid and one dimensional block
__global__ void imageExp(float *img, const int size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < size)
    {
        img[x] = expf(img[x]);
    }
}

// use one dimensional grid and one dimensional block
__global__ void imageSub(float *result, float *img1, float *img2, const int size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < size)
    {
        result[x] = img1[x] - img2[x];
    }
}

// use two dimensional grid and two dimensional block
__global__ void directionTop(const float *img, const float *weights,
                             const int n, const int group, const int column, const int row, float *result)
{
    float weight = weights[n + 2 - group - 1];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    int paddedR = r + group;
    int paddedC = c + n;
    int pColumn = column + n * 2;

    if (c < column && r < row)
    {
        result[r * column + c] +=
            weight * img[paddedR * pColumn + paddedC];
    }

    return;
}

// use two dimensional grid and two dimensional block
__global__ void directionBottom(const float *img, const float *weights,
                                const int n, const int group, const int column, const int row, float *result)
{
    float weight = weights[group];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    int paddedR = r + n - 1 + group;
    int paddedC = c + n;
    int pColumn = column + n * 2;

    if (c < column && r < row)
    {
        result[r * column + c] +=
            weight * img[paddedR * pColumn + paddedC];
    }

    return;
}

// use two dimensional grid and two dimensional block
__global__ void directionLeft(const float *img, const float *weights,
                              const int n, const int group, const int column, const int row, float *result)
{
    float weight = weights[n + 2 - group - 1];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    int paddedR = r + n;
    int paddedC = c + group;
    int pColumn = column + n * 2;

    if (c < column && r < row)
    {
        result[r * column + c] +=
            weight * img[paddedR * pColumn + paddedC];
    }

    return;
}

// use two dimensional grid and two dimensional block
__global__ void directionRight(const float *img, const float *weights,
                               const int n, const int group, const int column, const int row, float *result)
{
    float weight = weights[group];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    int paddedR = r + n;
    int paddedC = c + n - 1 + group;
    int pColumn = column + n * 2;

    if (c < column && r < row)
    {
        result[r * column + c] +=
            weight * img[paddedR * pColumn + paddedC];
    }

    return;
}
__global__ void directionLeftBottom(const float *img, const float *weights,
                                    const int n, const int group, const int column, const int row, float *result)
{
    float weight = weights[group];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    int paddedR = r + n + group - 1;
    int paddedC = c + n - group + 1;
    int pColumn = column + n * 2;

    if (c < column && r < row)
    {
        result[r * column + c] +=
            weight * img[paddedR * pColumn + paddedC];
    }

    return;
}
__global__ void directionRightTop(const float *img, const float *weights,
                                  const int n, const int group, const int column, const int row, float *result)
{
    float weight = weights[group];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    int paddedR = r + n + 1 - group;
    int paddedC = c + n - 1 + group;
    int pColumn = column + n * 2;

    if (c < column && r < row)
    {
        result[r * column + c] +=
            weight * img[paddedR * pColumn + paddedC];
    }

    return;
}
__global__ void directionLeftTop(const float *img, const float *weights,
                                 const int n, const int group, const int column, const int row, float *result)
{
    float weight = weights[group];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    int paddedR = r + n + 1 - group;
    int paddedC = c + n + 1 - group;
    int pColumn = column + n * 2;

    if (c < column && r < row)
    {
        result[r * column + c] +=
            weight * img[paddedR * pColumn + paddedC];
    }

    return;
}
__global__ void directionRightBottom(const float *img, const float *weights,
                                     const int n, const int group, const int column, const int row, float *result)
{
    float weight = weights[group];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    int paddedR = r + n - 1 + group;
    int paddedC = c + n - 1 + group;
    int pColumn = column + n * 2;

    if (c < column && r < row)
    {
        result[r * column + c] +=
            weight * img[paddedR * pColumn + paddedC];
    }

    return;
}

// use one dimensional grid and one dimensional block
__global__ void computeD_l(float *d_D_l, float *d_D_l_8, const int size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < size)
    {
        float *base = d_D_l_8;
        float *base1 = base + size;
        float *base2 = base1 + size;
        float *base3 = base2 + size;
        float *base4 = base3 + size;
        float *base5 = base4 + size;
        float *base6 = base5 + size;
        float *base7 = base6 + size;

        d_D_l[x] = abs(*(base + x)) + abs(*(base1 + x)) +
                   abs(*(base2 + x)) + abs(*(base3 + x)) + abs(*(base4 + x)) +
                   abs(*(base5 + x)) + abs(*(base6 + x)) + abs(*(base7 + x));
    }
}

void cal8DerivCuda(const float *d_vchannelImg, float *d_D, float *d_D8,
                   const float *d_mask, const int row, const int column)
{
    int size = row * column;
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((column + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (row + threadsPerBlock.y - 1) / threadsPerBlock.y);

    omp_set_num_threads(8);
#pragma omp parallel sections
    {
#pragma omp section
        {
            for (int i = 0; i < opt_.N; i++)
            {
                directionTop<<<numBlocks, threadsPerBlock>>>(d_vchannelImg, d_mask, opt_.N - 2,
                                                             i, column, row, d_D8);
            }
        }

#pragma omp section
        {
            for (int i = 0; i < opt_.N; i++)
            {
                directionBottom<<<numBlocks, threadsPerBlock>>>(d_vchannelImg, d_mask, opt_.N - 2,
                                                                i, column, row, d_D8 + size * 1);
            }
        }
#pragma omp section
        {
            for (int i = 0; i < opt_.N; i++)
            {
                directionLeft<<<numBlocks, threadsPerBlock>>>(d_vchannelImg, d_mask, opt_.N - 2,
                                                              i, column, row, d_D8 + size * 2);
            }
        }
#pragma omp section
        {
            for (int i = 0; i < opt_.N; i++)
            {
                directionRight<<<numBlocks, threadsPerBlock>>>(d_vchannelImg, d_mask, opt_.N - 2,
                                                               i, column, row, d_D8 + size * 3);
            }
        }
#pragma omp section
        {
            for (int i = 0; i < opt_.N; i++)
            {
                directionLeftBottom<<<numBlocks, threadsPerBlock>>>(d_vchannelImg, d_mask, opt_.N - 2,
                                                                    i, column, row, d_D8 + size * 4);
            }
        }
#pragma omp section
        {
            for (int i = 0; i < opt_.N; i++)
            {
                directionRightTop<<<numBlocks, threadsPerBlock>>>(d_vchannelImg, d_mask, opt_.N - 2,
                                                                  i, column, row, d_D8 + size * 5);
            }
        }
#pragma omp section
        {
            for (int i = 0; i < opt_.N; i++)
            {
                directionLeftTop<<<numBlocks, threadsPerBlock>>>(d_vchannelImg, d_mask, opt_.N - 2,
                                                                 i, column, row, d_D8 + size * 6);
            }
        }
#pragma omp section
        {
            for (int i = 0; i < opt_.N; i++)
            {
                directionRightBottom<<<numBlocks, threadsPerBlock>>>(d_vchannelImg, d_mask, opt_.N - 2,
                                                                     i, column, row, d_D8 + size * 7);
            }
        }
    }

    // compute D_l
    dim3 thread{64, 1};
    dim3 block{(size + thread.x - 1) / thread.x, 1};
    computeD_l<<<block, thread>>>(d_D, d_D8, size);
}

// use one dimensional grid and one dimensional block
__global__ void arrayPower(const float *img, float *result, const float power, const int size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x > size)
    {
        return;
    }

    if (power < 0)
    {
        result[x] = 1 / (pow(img[x], abs(power)));
    }
    else
    {
        result[x] = pow(img[x], power);
    }
}

// use two dimensional grid and two dimensional block
__global__ void secondTerm(const float *img, float *result, const float power,
                           const float alpha, const int row, const int column, const int n)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    if (r > row || c > column)
    {
        return;
    }

    int idx1 = r * column + c;
    int idx2 = (r + n) * (column + 2 * n) + c + n;
    if (power < 0)
    {
        result[idx1] = (alpha / (pow(abs(img[idx2]), abs(power)))) * img[idx2];
    }
    else
    {
        result[idx1] = alpha * pow(abs(img[idx2]), power) * img[idx2];
    }
}

// use two dimensional grid and two dimensional block
__global__ void computeTemp(const float *D_l_power, const float *D_l_8, const float *D_ls_power,
                            const float *D_ls_8, const float *second_term, const float alpha, const int row,
                            const int column, const int n, float *result)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    if ((c < column) && (r < row))
    {
        int idx1 = r * column + c;
        int idx2 = (r + n) * (column + 2 * n) + c + n;

        result[idx2] = D_l_power[idx1] * D_l_8[idx1] + second_term[idx1] +
                       alpha * D_ls_power[idx1] * D_ls_8[idx1];
    }
}

// use one dimensional grid and one dimensional block
__global__ void addEightTerms(float *eight_terms, const float *temp, const int size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < size)
    {
        eight_terms[x] += temp[x];
    }
}

// use one dimensional grid and one dimensional block
__global__ void computeSumk(float *sumk, float *eight_terms,
                            const float factor, const int size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < size)
    {
        sumk[x] += factor * eight_terms[x];
    }
}

// use one dimensional grid and one dimensional block
__global__ void computeResults(float *results, float *sumk, const float factor, const int size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < size)
    {
        results[x] = factor * sumk[x];
    }
}

// use one dimensional grid and one dimensional block
__global__ void horizontalPad(float *img, const int row, const int column, const int n)
{
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int paddedColumn = column + 2 * n;
    if (r < row)
    {
        // left
        for (int c = 0; c < n; c++)
        {
            int idx1 = (r + n) * paddedColumn + n - 1 - c;
            int idx2 = (r + n) * paddedColumn + n + c;
            img[idx1] = img[idx2];
        }

        // right
        for (int c = 0; c < n; c++)
        {
            int idx1 = (r + n) * paddedColumn + c + n + column;
            int idx2 = (r + n) * paddedColumn + n + column - 1 - c;
            img[idx1] = img[idx2];
        }
    }
}

// use one dimensional grid and one dimensional block
__global__ void verticalPad(float *img, const int row, const int column, const int n)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int paddedColumn = column + 2 * n;

    if (c < paddedColumn)
    {
        // top
        for (int r = 0; r < n; r++)
        {
            int idx1 = (n - 1 - r) * paddedColumn + c;
            int idx2 = (n + r) * paddedColumn + c;

            img[idx1] = img[idx2];
        }

        // bottom

        for (int r = 0; r < n; r++)
        {
            int idx1 = (n + row + r) * paddedColumn + c;
            int idx2 = (n + row - 1 - r) * paddedColumn + c;
            img[idx1] = img[idx2];
        }
    }
}

// note that the image itself is large enough to hold the padded imge
void symetricPad(float *img, const int row, const int column, const int n)
{

    float *d_img;
    printIfCudaFailed(hipMalloc(&d_img, sizeof(float) * (row + 2 * n) * (column + 2 * n)));
    printIfCudaFailed(hipMemcpy(d_img, img, sizeof(float) * (row + 2 * n) * (column + 2 * n),
                                 hipMemcpyHostToDevice));

    dim3 threadBlock{64, 1};
    dim3 blockGrid{(row + threadBlock.x - 1) / threadBlock.x, 1};
    horizontalPad<<<blockGrid, threadBlock>>>(d_img, row, column, n);

    dim3 blockGrid2{(column + 2 * n + threadBlock.x - 1) / threadBlock.x, 1};
    verticalPad<<<threadBlock, blockGrid2>>>(d_img, row, column, n);

    printIfCudaFailed(hipMemcpy(img, d_img, sizeof(float) * (row + 2 * n) * (column + 2 * n),
                                 hipMemcpyDeviceToHost));
    printIfCudaFailed(hipFree(d_img));
}

// note that the image itself is large enough to hold the padded imge
void symetricPadDevice(float *d_img, const int row, const int column, const int n)
{
    dim3 threadBlock{64, 1};
    dim3 blockGrid{(row + threadBlock.x - 1) / threadBlock.x, 1};
    horizontalPad<<<blockGrid, threadBlock>>>(d_img, row, column, n);

    dim3 blockGrid2{(column + 2 * n + threadBlock.x - 1) / threadBlock.x, 1};
    verticalPad<<<threadBlock, blockGrid2>>>(d_img, row, column, n);
}

void fstDiffCuda(float *d_img, float *d_result, const float *d_mask, const int layer,
                 const int row, const int column)
{
    int n = opt_.N - 2;
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((column + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (row + threadsPerBlock.y - 1) / threadsPerBlock.y);
    switch (layer)
    {
    case 0:
        for (int i = 0; i < 4; i++)
        {
            directionTop<<<numBlocks, threadsPerBlock>>>(d_img, d_mask, n, i, column, row, d_result);
        }

        break;
    case 1:
        for (int i = 0; i < 4; i++)
        {
            directionBottom<<<numBlocks, threadsPerBlock>>>(d_img, d_mask, n, i, column, row, d_result);
        }

        break;
    case 2:
        for (int i = 0; i < 4; i++)
        {
            directionLeft<<<numBlocks, threadsPerBlock>>>(d_img, d_mask, n, i, column, row, d_result);
        }

        break;
    case 3:
        for (int i = 0; i < 4; i++)
        {
            directionRight<<<numBlocks, threadsPerBlock>>>(d_img, d_mask, n, i, column, row, d_result);
        }

        break;
    case 4:
        for (int i = 0; i < 4; i++)
        {
            directionLeftBottom<<<numBlocks, threadsPerBlock>>>(d_img, d_mask, n, i, column, row, d_result);
        }

        break;
    case 5:
        for (int i = 0; i < 4; i++)
        {
            directionRightTop<<<numBlocks, threadsPerBlock>>>(d_img, d_mask, n, i, column, row, d_result);
        }

        break;
    case 6:
        for (int i = 0; i < 4; i++)
        {
            directionLeftTop<<<numBlocks, threadsPerBlock>>>(d_img, d_mask, n, i, column, row, d_result);
        }

        break;
    case 7:
        for (int i = 0; i < 4; i++)
        {
            directionRightBottom<<<numBlocks, threadsPerBlock>>>(d_img, d_mask, n, i, column, row, d_result);
        }

        break;
    default:
        break;
    }
}

void utTestPrintDeviceVector(float *d_img, string infor, int size, int limit, int column)
{
    vector<float> img(size, 0);
    printIfCudaFailed(hipMemcpy(&(img[0]), d_img, sizeof(float) * size, hipMemcpyDeviceToHost));

    cout << infor << endl;
    for (int i = 0; i < limit; i++)
    {
        cout << img[i]
             << "\t";
        if ((i + 1) % column == 0)
        {
            cout << "\n";
        }
    }
    cout << endl;
}

// fuse eight calls to computeTemp to one kernel, use twe dimensional grid and twe dimensional block
__global__ void compute8Temp(const float *D_l_power, float *D_l_8, const float *D_ls_power,
                             float *D_ls_8, const float *second_term, const float alpha, const int row,
                             const int column, const int n, float *result)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    if (c > column || r > row)
    {
        return;
    }

    __shared__ float D_l_power_temp[32 * 32];
    __shared__ float D_ls_power_temp[32 * 32];
    __shared__ float second_term_temp[32 * 32];

    // copy data from memory to shared memory
    int idx = r * column + c;
    int tempIdx = threadIdx.y * 32 + threadIdx.x;
    D_l_power_temp[tempIdx] = D_l_power[idx];
    __syncthreads();

    D_ls_power_temp[tempIdx] = D_ls_power[idx];
    __syncthreads();

    second_term_temp[tempIdx] = second_term[idx];
    __syncthreads();

    // compute results
    float *D_l_8_base = D_l_8;
    float *D_ls_8_base = D_ls_8;
    float *result_base = result;
    int idx2 = (r + n) * (column + 2 * n) + c + n;
    int size = row * column;
    int paddedSize = (row + 2 * n) * (column + 2 * n);

    // layer 0
    result_base[idx2] = D_l_power_temp[tempIdx] * D_l_8[idx] + second_term_temp[tempIdx] +
                        alpha * D_ls_power_temp[tempIdx] * D_ls_8[idx];
    D_l_8_base += size;
    D_ls_8_base += size;
    result_base += paddedSize;

    __syncthreads();
    // layer 1
    result_base[idx2] = D_l_power_temp[tempIdx] * D_l_8[idx] + second_term_temp[tempIdx] +
                        alpha * D_ls_power_temp[tempIdx] * D_ls_8[idx];
    D_l_8_base += size;
    D_ls_8_base += size;
    result_base += paddedSize;

    __syncthreads();
    // layer 2
    result_base[idx2] = D_l_power_temp[tempIdx] * D_l_8[idx] + second_term_temp[tempIdx] +
                        alpha * D_ls_power_temp[tempIdx] * D_ls_8[idx];
    D_l_8_base += size;
    D_ls_8_base += size;
    result_base += paddedSize;

    __syncthreads();
    // layer 3
    result_base[idx2] = D_l_power_temp[tempIdx] * D_l_8[idx] + second_term_temp[tempIdx] +
                        alpha * D_ls_power_temp[tempIdx] * D_ls_8[idx];
    D_l_8_base += size;
    D_ls_8_base += size;
    result_base += paddedSize;

    __syncthreads();

    // layer 4
    result_base[idx2] = D_l_power_temp[tempIdx] * D_l_8[idx] + second_term_temp[tempIdx] +
                        alpha * D_ls_power_temp[tempIdx] * D_ls_8[idx];
    D_l_8_base += size;
    D_ls_8_base += size;
    result_base += paddedSize;

    __syncthreads();

    // layer 5
    result_base[idx2] = D_l_power_temp[tempIdx] * D_l_8[idx] + second_term_temp[tempIdx] +
                        alpha * D_ls_power_temp[tempIdx] * D_ls_8[idx];
    D_l_8_base += size;
    D_ls_8_base += size;
    result_base += paddedSize;

    __syncthreads();

    // layer 6
    result_base[idx2] = D_l_power_temp[tempIdx] * D_l_8[idx] + second_term_temp[tempIdx] +
                        alpha * D_ls_power_temp[tempIdx] * D_ls_8[idx];
    D_l_8_base += size;
    D_ls_8_base += size;
    result_base += paddedSize;

    __syncthreads();

    // layer 7
    result_base[idx2] = D_l_power_temp[tempIdx] * D_l_8[idx] + second_term_temp[tempIdx] +
                        alpha * D_ls_power_temp[tempIdx] * D_ls_8[idx];
}

// fuse eight calls to addEightTerms to one kernel, use one dimensional grid and one dimensional block
__global__ void add8EightTerms(float *eight_terms, float *temp, const int size)
{
    __shared__ float eight_terms_temp[64];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x > size)
    {
        return;
    }

    float *temp_base = temp;

    // layer 0
    eight_terms_temp[threadIdx.x] = temp_base[x];
    if (threadIdx.x == 0)
    {
        temp_base += size;
    }
    __syncthreads();

    // layer 1
    eight_terms_temp[threadIdx.x] += temp_base[x];
    if (threadIdx.x == 0)
    {
        temp_base += size;
    }
    __syncthreads();

    // layer 2
    eight_terms_temp[threadIdx.x] += temp_base[x];
    if (threadIdx.x == 0)
    {
        temp_base += size;
    }
    __syncthreads();

    // layer 3
    eight_terms_temp[threadIdx.x] += temp_base[x];
    if (threadIdx.x == 0)
    {
        temp_base += size;
    }
    __syncthreads();

    // layer 4
    eight_terms_temp[threadIdx.x] += temp_base[x];
    if (threadIdx.x == 0)
    {
        temp_base += size;
    }
    __syncthreads();

    // layer 5
    eight_terms_temp[threadIdx.x] += temp_base[x];
    if (threadIdx.x == 0)
    {
        temp_base += size;
    }
    __syncthreads();

    // layer 6
    eight_terms_temp[threadIdx.x] += temp_base[x];
    if (threadIdx.x == 0)
    {
        temp_base += size;
    }
    __syncthreads();

    // layer 7
    eight_terms_temp[threadIdx.x] += temp_base[x];
    __syncthreads();

    // wirte results back to memory
    eight_terms[x] = eight_terms_temp[threadIdx.x];
}

// fuse compute8EightTerms and compute sumk to one kernel to reduce global memory access
__global__ void fusedKernel2(float *sumk, float *temp_results, const float factor, const int size)
{
    __shared__ float eight_terms_temp[64];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x > size)
    {
        return;
    }

    float *temp_base = temp_results;

    // layer 0
    eight_terms_temp[threadIdx.x] = temp_base[x];
    temp_base += size;

    __syncthreads();

    // layer 1
    eight_terms_temp[threadIdx.x] += temp_base[x];
    temp_base += size;

    __syncthreads();

    // layer 2
    eight_terms_temp[threadIdx.x] += temp_base[x];
    temp_base += size;

    __syncthreads();

    // layer 3
    eight_terms_temp[threadIdx.x] += temp_base[x];
    temp_base += size;

    __syncthreads();

    // layer 4
    eight_terms_temp[threadIdx.x] += temp_base[x];
    temp_base += size;

    __syncthreads();

    // layer 5
    eight_terms_temp[threadIdx.x] += temp_base[x];
    temp_base += size;

    __syncthreads();

    // layer 6
    eight_terms_temp[threadIdx.x] += temp_base[x];
    temp_base += size;

    __syncthreads();

    // layer 7
    eight_terms_temp[threadIdx.x] += temp_base[x];
    __syncthreads();

    // wirte results back to memory
    sumk[x] += eight_terms_temp[threadIdx.x] * factor;
}

void computeP(float *d_l_curr, float *d_s, float *d_mask, float *d_ls, float *d_results,
              float *d_D_l, float *d_D_ls, float *d_D_l_8, float *d_D_ls_8,
              float *d_D_l_power, float *d_D_ls_power,
              float *d_second_term, float *d_temp_pad, float *d_temp_results,
              float *d_sumk, const int row, const int column)
{
    int n = opt_.N - 2;
    int size = row * column;
    int paddedSize = (row + 2 * n) * (column + 2 * n);

    dim3 threadBlock{64, 1};
    dim3 blockGrid{(paddedSize + threadBlock.x - 1) / threadBlock.x, 1};
    imageSub<<<blockGrid, threadBlock>>>(d_ls, d_l_curr, d_s, paddedSize);

    hipMemset(d_D_l_8, 0, 8 * size * sizeof(float));
    hipMemset(d_D_ls_8, 0, 8 * size * sizeof(float));
    dim3 threadBlock1{64, 1};
    dim3 blockGrid1{(size + threadBlock1.x - 1) / threadBlock1.x, 1};

    symetricPadDevice(d_l_curr, row, column, n);
    cal8DerivCuda(d_l_curr, d_D_l, d_D_l_8, d_mask, row, column);

    symetricPadDevice(d_ls, row, column, n);
    cal8DerivCuda(d_ls, d_D_ls, d_D_ls_8, d_mask, row, column);

    imageRelpaceWith<<<blockGrid1, threadBlock1>>>(d_D_l, size, opt_.epsilon_1);
    imageRelpaceWith<<<blockGrid1, threadBlock1>>>(d_D_ls, size, opt_.epsilon_1);

    // 对 k 进行遍历，将求和符号逐 k 累加到 sum_k
    hipMemset(d_sumk, 0, size * sizeof(float));
    for (int k = 0; k < 2; k++)
    {
        float prod_tau = 1.;
        for (int tau = 1; tau <= 2 * k; tau++)
        {
            prod_tau *= (opt_.v_2 - tau + 1);
        }

        // 计算在当前 k 值的情况下，八个方向的偏微分的差分的和
        dim3 thread{32, 32};
        dim3 block{(column + thread.x - 1) / thread.x,
                   (row + thread.y - 1) / thread.y};

        arrayPower<<<blockGrid1, threadBlock1>>>(d_D_l, d_D_l_power, opt_.v_2 - 2 * k - 2, size);

        arrayPower<<<blockGrid1, threadBlock1>>>(d_D_ls, d_D_ls_power, opt_.v_2 - 2 * k - 2, size);

        secondTerm<<<block, thread>>>(d_ls, d_second_term, opt_.v_2 - 2 * k - 2,
                                      opt_.alpha_1, row, column, n);

        // iterate through eight directions
        compute8Temp<<<block, thread>>>(d_D_l_power, d_D_l_8, d_D_ls_power,
                                        d_D_ls_8, d_second_term,
                                        opt_.alpha_2, row, column, n, d_temp_pad);

        symetricPadDevice(d_temp_pad, row, column, n);
        symetricPadDevice(d_temp_pad + paddedSize, row, column, n);
        symetricPadDevice(d_temp_pad + paddedSize * 2, row, column, n);
        symetricPadDevice(d_temp_pad + paddedSize * 3, row, column, n);
        symetricPadDevice(d_temp_pad + paddedSize * 4, row, column, n);
        symetricPadDevice(d_temp_pad + paddedSize * 5, row, column, n);
        symetricPadDevice(d_temp_pad + paddedSize * 6, row, column, n);
        symetricPadDevice(d_temp_pad + paddedSize * 7, row, column, n);

        hipMemset(d_temp_results, 0, size * sizeof(float));
#pragma omp parallel for
        for (int layer = 0; layer < 8; layer++)
        {
            fstDiffCuda(d_temp_pad + layer * size, d_temp_results + layer * size, d_mask, layer, row, column);
        }

        utTestPrintDeviceVector(d_temp_results, "d_temp_results0", size, size, column);
        utTestPrintDeviceVector(d_temp_results + size, "d_temp_results1", size, size, column);
        utTestPrintDeviceVector(d_temp_results + size * 2, "d_temp_results2", size, size, column);
        utTestPrintDeviceVector(d_temp_results + size * 3, "d_temp_results3", size, size, column);
        utTestPrintDeviceVector(d_temp_results + size * 4, "d_temp_results4", size, size, column);
        utTestPrintDeviceVector(d_temp_results + size * 5, "d_temp_results5", size, size, column);
        utTestPrintDeviceVector(d_temp_results + size * 6, "d_temp_results6", size, size, column);
        utTestPrintDeviceVector(d_temp_results + size * 7, "d_temp_results7", size, size, column);

        fusedKernel2<<<blockGrid1, threadBlock1>>>(d_sumk, d_temp_results, prod_tau / tgammaf(2 * k + 1), size);
    }

    // do fractional derivative
    computeResults<<<blockGrid1, threadBlock1>>>(d_results, d_sumk,
                                                 -tgammaf(1 - opt_.v_1) / tgammaf(-opt_.v_1) / tgammaf(-opt_.v_3),
                                                 size);
}

// use two dimensional grid and two dimensional block
__global__ void computeDealtaL(float *Delta_l, float *p_l, float *l_curr, const int row, const int column,
                               const int n, const float v3, const float Delta_t, const float mu)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    if (c > column || r > row)
    {
        return;
    }

    int idx1 = r * column + c;
    int idx2 = (r + n) * (column + 2 * n) + c + n;

    if (v3 > 0)
    {
        Delta_l[idx1] = p_l[idx1] * pow(Delta_t, v3) -
                        2 * mu / tgammaf(3 - v3) * (Delta_l[idx1] * Delta_l[idx1]) * 1 / (pow(l_curr[idx2], v3));
    }
    else
    {
        Delta_l[idx1] = p_l[idx1] * pow(Delta_t, v3) -
                        2 * mu / tgammaf(3 - v3) * (Delta_l[idx1] * Delta_l[idx1]) * pow(l_curr[idx2], -v3);
    }
}

// fuse computeDealtaL, update_l_curr, imageRelpaceWith
__global__ void fusedKernel1(float *d_l_curr, float *d_dealt_l, float *d_p_l,
                             const int row, const int column, const int n,
                             const float computeConst1, const float computeConst2,
                             const float v3, const float threashold)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    if (c > column || r > row)
    {
        return;
    }

    int idx1 = r * column + c;
    int idx2 = (r + n) * (column + 2 * n) + c + n;

    // read data from memory
    float p_l_ = d_p_l[idx1];
    float Delta_l_ = d_dealt_l[idx2];
    float l_curr_ = d_l_curr[idx1];

    // compute
    float temp = p_l_ * computeConst1 - computeConst2 * Delta_l_ * Delta_l_ * 1 / (pow(l_curr_, v3));
    float temp1 = (l_curr_ + temp);
    if (temp1 < threashold)
    {
        temp1 = threashold;
    }

    // store results back to memroy
    d_dealt_l[idx2] = temp;
    d_l_curr[idx1] = temp1;
}

void franctionalRetinexCuda(float *l_curr, const float *s, const float *c, const int row, const int column)
{
    float *d_l_curr, *d_s, *d_c, *d_ls;
    float *d_p_l, *d_delta_l;
    float *d_D_l, *d_D_l_s;
    // used in computeP for to store temp results
    float *d_D_l_8, *d_D_ls_8, *d_D_l_power, *d_D_ls_power, *d_second_term,
        *d_temp_pad, *d_temp_results, *d_sumk;

    int size = row * column;
    int paddedSize = (row + 2 * (opt_.N - 2)) * (column + 2 * (opt_.N - 2));
    int imgSize = sizeof(float) * size;
    int paddedImgSize = sizeof(float) * paddedSize;
    int templateSize = sizeof(float) * opt_.N;

    // prepare memory on device
    printIfCudaFailed(hipMalloc(&d_l_curr, paddedImgSize));
    printIfCudaFailed(hipMalloc(&d_s, paddedImgSize));
    printIfCudaFailed(hipMalloc(&d_ls, paddedImgSize));
    printIfCudaFailed(hipMalloc(&d_c, templateSize));
    printIfCudaFailed(hipMalloc(&d_p_l, imgSize));
    printIfCudaFailed(hipMalloc(&d_delta_l, imgSize));
    printIfCudaFailed(hipMalloc(&d_D_l, imgSize));
    printIfCudaFailed(hipMalloc(&d_D_l_s, imgSize));
    printIfCudaFailed(hipMalloc(&d_D_l_8, 8 * imgSize));
    printIfCudaFailed(hipMalloc(&d_D_ls_8, 8 * imgSize));
    printIfCudaFailed(hipMalloc(&d_D_l_power, imgSize));
    printIfCudaFailed(hipMalloc(&d_D_ls_power, imgSize));
    printIfCudaFailed(hipMalloc(&d_second_term, imgSize));
    printIfCudaFailed(hipMalloc(&d_temp_pad, 8 * paddedImgSize));
    printIfCudaFailed(hipMalloc(&d_temp_results, 8 * imgSize));
    printIfCudaFailed(hipMalloc(&d_sumk, imgSize));

    printIfCudaFailed(hipMemcpy(d_l_curr, l_curr, paddedImgSize, hipMemcpyHostToDevice));
    printIfCudaFailed(hipMemcpy(d_s, s, paddedImgSize, hipMemcpyHostToDevice));
    printIfCudaFailed(hipMemcpy(d_c, c, templateSize, hipMemcpyHostToDevice));

    int n = opt_.N - 2;
    dim3 threadBlock{32, 32};
    dim3 blockGrid{(column + threadBlock.x - 1) / threadBlock.x, (row + threadBlock.y - 1) / threadBlock.y};
    dim3 Block{64, 1};
    dim3 Grid{(paddedSize + Block.x - 1) / Block.x, 1};

    for (int t = 0; t < 1; t++)
    {
        computeP(d_l_curr, d_s, d_c, d_ls, d_p_l, d_D_l, d_D_l_s, d_D_l_8, d_D_ls_8,
                 d_D_l_power, d_D_ls_power, d_second_term,
                 d_temp_pad, d_temp_results, d_sumk, row, column);

        utTestPrintDeviceVector(d_p_l, "d_p_l", size, size, column);

        // debug
        cout << "computeP " << endl;
        fusedKernel1<<<blockGrid, threadBlock>>>(d_l_curr, d_delta_l, d_p_l, row, column, n,
                                                 pow(opt_.Delta_t, opt_.v_3),
                                                 2 * opt_.mu / tgammaf(3 - opt_.v_3),
                                                 opt_.v_3, opt_.epsilon_2);
    }

    // L = np.exp(l_curr)
    imageExp<<<Grid, Block>>>(d_l_curr, paddedSize);

    printIfCudaFailed(hipMemcpy(l_curr, d_l_curr, paddedImgSize, hipMemcpyDeviceToHost));

    // free memory
    printIfCudaFailed(hipFree(d_l_curr));
    printIfCudaFailed(hipFree(d_s));
    printIfCudaFailed(hipFree(d_c));
    printIfCudaFailed(hipFree(d_ls));
    printIfCudaFailed(hipFree(d_p_l));
    printIfCudaFailed(hipFree(d_delta_l));
    printIfCudaFailed(hipFree(d_D_l));
    printIfCudaFailed(hipFree(d_D_l_s));
    printIfCudaFailed(hipFree(d_D_l_8));
    printIfCudaFailed(hipFree(d_D_ls_8));
    printIfCudaFailed(hipFree(d_D_l_power));
    printIfCudaFailed(hipFree(d_D_ls_power));
    printIfCudaFailed(hipFree(d_second_term));
    printIfCudaFailed(hipFree(d_temp_pad));
    printIfCudaFailed(hipFree(d_temp_results));
    printIfCudaFailed(hipFree(d_sumk));
}

void utTestPadCuda(float *img, const int row, const int column, const int n)
{
    float *d_img;
    int paddedImgSize = sizeof(float) * (row + 2 * n) * (column + 2 * n);

    printIfCudaFailed(hipMalloc(&d_img, paddedImgSize));
    printIfCudaFailed(hipMemcpy(d_img, img, paddedImgSize, hipMemcpyHostToDevice));

    symetricPadDevice(d_img, row, column, n);

    printIfCudaFailed(hipMemcpy(img, d_img, paddedImgSize, hipMemcpyDeviceToHost));
    printIfCudaFailed(hipFree(d_img));
}

void utTestCal8DrivCuda(float *img, float *D_l, float *D_l_8, const float *mask,
                        const int row, const int column, const int n)
{
    int size = row * column;
    int paddedSize = (row + 2 * n) * (column + 2 * n);
    int imgSize = sizeof(float) * size;
    int paddedImgSize = sizeof(float) * paddedSize;

    float *d_img, *d_D_l, *d_D_l_8, *d_mask;
    printIfCudaFailed(hipMalloc(&d_img, paddedImgSize));
    printIfCudaFailed(hipMalloc(&d_D_l, imgSize));
    printIfCudaFailed(hipMalloc(&d_D_l_8, 8 * imgSize));
    printIfCudaFailed(hipMalloc(&d_mask, sizeof(float) * (n + 2)));

    printIfCudaFailed(hipMemcpy(d_img, img, paddedImgSize, hipMemcpyHostToDevice));
    printIfCudaFailed(hipMemcpy(d_mask, mask, sizeof(float) * (n + 2), hipMemcpyHostToDevice));
    printIfCudaFailed(hipMemset(d_D_l, 0, imgSize));

    symetricPadDevice(d_img, row, column, n);
    cal8DerivCuda(d_img, d_D_l, d_D_l_8, d_mask, row, column);

    printIfCudaFailed(hipMemcpy(D_l, d_D_l, imgSize, hipMemcpyDeviceToHost));
    printIfCudaFailed(hipMemcpy(D_l_8, d_D_l_8, 8 * imgSize, hipMemcpyDeviceToHost));

    printIfCudaFailed(hipFree(d_img));
    printIfCudaFailed(hipFree(d_D_l));
    printIfCudaFailed(hipFree(d_D_l_8));
    printIfCudaFailed(hipFree(d_mask));
}